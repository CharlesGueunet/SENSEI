#include "hip/hip_runtime.h"
//============================================================================
//  Copyright (c) Kitware, Inc.
//  All rights reserved.
//  See LICENSE.txt for details.
//
//  This software is distributed WITHOUT ANY WARRANTY; without even
//  the implied warranty of MERCHANTABILITY or FITNESS FOR A PARTICULAR
//  PURPOSE.  See the above copyright notice for more information.
//============================================================================
#include <svtkm/testing/Testing.h>

#include <svtkm/cont/cuda/DeviceAdapterCuda.h>

#include <svtkm/exec/FunctorBase.h>
#include <svtkm/exec/arg/BasicArg.h>
#include <svtkm/exec/arg/ThreadIndicesBasic.h>
#include <svtkm/exec/cuda/internal/TaskStrided.h>

#include <svtkm/StaticAssert.h>

#include <svtkm/internal/FunctionInterface.h>
#include <svtkm/internal/Invocation.h>

#if defined(SVTKM_MSVC)
#pragma warning(push)
#pragma warning(disable : 4068) //unknown pragma
#endif

#if defined(__NVCC__) && defined(__CUDACC_VER_MAJOR__)
// Disable warning "declared but never referenced"
// This file produces several false-positive warnings
// Eg: TestExecObject::TestExecObject, MyOutputToInputMapPortal::Get,
//     TestWorkletProxy::operator()
#pragma push
#pragma diag_suppress 177
#endif

namespace
{

struct TestExecObject
{
  SVTKM_EXEC_CONT
  TestExecObject(svtkm::exec::cuda::internal::ArrayPortalFromThrust<svtkm::Id> portal)
    : Portal(portal)
  {
  }

  svtkm::exec::cuda::internal::ArrayPortalFromThrust<svtkm::Id> Portal;
};

struct MyOutputToInputMapPortal
{
  using ValueType = svtkm::Id;
  SVTKM_EXEC_CONT
  svtkm::Id Get(svtkm::Id index) const { return index; }
};

struct MyVisitArrayPortal
{
  using ValueType = svtkm::IdComponent;
  SVTKM_EXEC_CONT
  svtkm::IdComponent Get(svtkm::Id) const { return 1; }
};

struct MyThreadToOutputMapPortal
{
  using ValueType = svtkm::Id;
  SVTKM_EXEC_CONT
  svtkm::Id Get(svtkm::Id index) const { return index; }
};

struct TestFetchTagInput
{
};
struct TestFetchTagOutput
{
};

// Missing TransportTag, but we are not testing that so we can leave it out.
struct TestControlSignatureTagInput
{
  using FetchTag = TestFetchTagInput;
};
struct TestControlSignatureTagOutput
{
  using FetchTag = TestFetchTagOutput;
};

} // anonymous namespace

namespace svtkm
{
namespace exec
{
namespace arg
{

template <>
struct Fetch<TestFetchTagInput,
             svtkm::exec::arg::AspectTagDefault,
             svtkm::exec::arg::ThreadIndicesBasic,
             TestExecObject>
{
  using ValueType = svtkm::Id;

  SVTKM_EXEC
  ValueType Load(const svtkm::exec::arg::ThreadIndicesBasic& indices,
                 const TestExecObject& execObject) const
  {
    return execObject.Portal.Get(indices.GetInputIndex()) + 10 * indices.GetInputIndex();
  }

  SVTKM_EXEC
  void Store(const svtkm::exec::arg::ThreadIndicesBasic&, const TestExecObject&, ValueType) const
  {
    // No-op
  }
};

template <>
struct Fetch<TestFetchTagOutput,
             svtkm::exec::arg::AspectTagDefault,
             svtkm::exec::arg::ThreadIndicesBasic,
             TestExecObject>
{
  using ValueType = svtkm::Id;

  SVTKM_EXEC
  ValueType Load(const svtkm::exec::arg::ThreadIndicesBasic&, const TestExecObject&) const
  {
    // No-op
    return ValueType();
  }

  SVTKM_EXEC
  void Store(const svtkm::exec::arg::ThreadIndicesBasic& indices,
             const TestExecObject& execObject,
             ValueType value) const
  {
    execObject.Portal.Set(indices.GetOutputIndex(), value + 20 * indices.GetOutputIndex());
  }
};
}
}
} // svtkm::exec::arg

namespace
{

using TestControlSignature = void(TestControlSignatureTagInput, TestControlSignatureTagOutput);
using TestControlInterface = svtkm::internal::FunctionInterface<TestControlSignature>;

using TestExecutionSignature1 = void(svtkm::exec::arg::BasicArg<1>, svtkm::exec::arg::BasicArg<2>);
using TestExecutionInterface1 = svtkm::internal::FunctionInterface<TestExecutionSignature1>;

using TestExecutionSignature2 = svtkm::exec::arg::BasicArg<2>(svtkm::exec::arg::BasicArg<1>);
using TestExecutionInterface2 = svtkm::internal::FunctionInterface<TestExecutionSignature2>;

using ExecutionParameterInterface =
  svtkm::internal::FunctionInterface<void(TestExecObject, TestExecObject)>;

using InvocationType1 = svtkm::internal::Invocation<ExecutionParameterInterface,
                                                   TestControlInterface,
                                                   TestExecutionInterface1,
                                                   1,
                                                   MyOutputToInputMapPortal,
                                                   MyVisitArrayPortal,
                                                   MyThreadToOutputMapPortal>;

using InvocationType2 = svtkm::internal::Invocation<ExecutionParameterInterface,
                                                   TestControlInterface,
                                                   TestExecutionInterface2,
                                                   1,
                                                   MyOutputToInputMapPortal,
                                                   MyVisitArrayPortal,
                                                   MyThreadToOutputMapPortal>;

template <typename TaskType>
static __global__ void ScheduleTaskStrided(TaskType task, svtkm::Id start, svtkm::Id end)
{

  const svtkm::Id index = blockIdx.x * blockDim.x + threadIdx.x;
  const svtkm::Id inc = blockDim.x * gridDim.x;
  if (index >= start && index < end)
  {
    task(index, end, inc);
  }
}

// Not a full worklet, but provides operators that we expect in a worklet.
struct TestWorkletProxy : svtkm::exec::FunctorBase
{
  SVTKM_EXEC
  void operator()(svtkm::Id input, svtkm::Id& output) const { output = input + 100; }

  SVTKM_EXEC
  svtkm::Id operator()(svtkm::Id input) const { return input + 200; }

  template <typename T,
            typename OutToInArrayType,
            typename VisitArrayType,
            typename ThreadToOutArrayType,
            typename InputDomainType,
            typename G>
  SVTKM_EXEC svtkm::exec::arg::ThreadIndicesBasic GetThreadIndices(
    const T& threadIndex,
    const OutToInArrayType& outToIn,
    const VisitArrayType& visit,
    const ThreadToOutArrayType& threadToOut,
    const InputDomainType&,
    const G& globalThreadIndexOffset) const
  {
    svtkm::Id outIndex = threadToOut.Get(threadIndex);
    return svtkm::exec::arg::ThreadIndicesBasic(
      threadIndex, outToIn.Get(outIndex), visit.Get(outIndex), outIndex, globalThreadIndexOffset);
  }
};

#define ERROR_MESSAGE "Expected worklet error."

// Not a full worklet, but provides operators that we expect in a worklet.
struct TestWorkletErrorProxy : svtkm::exec::FunctorBase
{
  SVTKM_EXEC
  void operator()(svtkm::Id, svtkm::Id) const { this->RaiseError(ERROR_MESSAGE); }

  template <typename T,
            typename OutToInArrayType,
            typename VisitArrayType,
            typename ThreadToOutArrayType,
            typename InputDomainType,
            typename G>
  SVTKM_EXEC svtkm::exec::arg::ThreadIndicesBasic GetThreadIndices(
    const T& threadIndex,
    const OutToInArrayType& outToIn,
    const VisitArrayType& visit,
    const ThreadToOutArrayType& threadToOut,
    const InputDomainType&,
    const G& globalThreadIndexOffset) const
  {
    svtkm::Id outIndex = threadToOut.Get(threadIndex);
    return svtkm::exec::arg::ThreadIndicesBasic(
      threadIndex, outToIn.Get(outIndex), visit.Get(outIndex), outIndex, globalThreadIndexOffset);
  }
};

// Check behavior of InvocationToFetch helper class.

SVTKM_STATIC_ASSERT(
  (std::is_same<svtkm::exec::internal::detail::
                  InvocationToFetch<svtkm::exec::arg::ThreadIndicesBasic, InvocationType1, 1>::type,
                svtkm::exec::arg::Fetch<TestFetchTagInput,
                                       svtkm::exec::arg::AspectTagDefault,
                                       svtkm::exec::arg::ThreadIndicesBasic,
                                       TestExecObject>>::type::value));

SVTKM_STATIC_ASSERT(
  (std::is_same<svtkm::exec::internal::detail::
                  InvocationToFetch<svtkm::exec::arg::ThreadIndicesBasic, InvocationType1, 2>::type,
                svtkm::exec::arg::Fetch<TestFetchTagOutput,
                                       svtkm::exec::arg::AspectTagDefault,
                                       svtkm::exec::arg::ThreadIndicesBasic,
                                       TestExecObject>>::type::value));

SVTKM_STATIC_ASSERT(
  (std::is_same<svtkm::exec::internal::detail::
                  InvocationToFetch<svtkm::exec::arg::ThreadIndicesBasic, InvocationType2, 0>::type,
                svtkm::exec::arg::Fetch<TestFetchTagOutput,
                                       svtkm::exec::arg::AspectTagDefault,
                                       svtkm::exec::arg::ThreadIndicesBasic,
                                       TestExecObject>>::type::value));

template <typename DeviceAdapter>
void TestNormalFunctorInvoke()
{
  std::cout << "Testing normal worklet invoke." << std::endl;

  svtkm::Id inputTestValues[3] = { 5, 5, 6 };

  svtkm::cont::ArrayHandle<svtkm::Id> input = svtkm::cont::make_ArrayHandle(inputTestValues, 3);
  svtkm::cont::ArrayHandle<svtkm::Id> output;

  svtkm::internal::FunctionInterface<void(TestExecObject, TestExecObject)> execObjects =
    svtkm::internal::make_FunctionInterface<void>(
      TestExecObject(input.PrepareForInPlace(DeviceAdapter())),
      TestExecObject(output.PrepareForOutput(3, DeviceAdapter())));

  std::cout << "  Try void return." << std::endl;
  TestWorkletProxy worklet;
  InvocationType1 invocation1(execObjects);

  using TaskTypes = typename svtkm::cont::DeviceTaskTypes<DeviceAdapter>;
  auto task1 = TaskTypes::MakeTask(worklet, invocation1, svtkm::Id());

  ScheduleTaskStrided<decltype(task1)><<<32, 256>>>(task1, 1, 2);
  hipDeviceSynchronize();
  input.SyncControlArray();
  output.SyncControlArray();

  SVTKM_TEST_ASSERT(inputTestValues[1] == 5, "Input value changed.");
  SVTKM_TEST_ASSERT(output.GetPortalConstControl().Get(1) == inputTestValues[1] + 100 + 30,
                   "Output value not set right.");

  std::cout << "  Try return value." << std::endl;

  execObjects = svtkm::internal::make_FunctionInterface<void>(
    TestExecObject(input.PrepareForInPlace(DeviceAdapter())),
    TestExecObject(output.PrepareForOutput(3, DeviceAdapter())));

  InvocationType2 invocation2(execObjects);

  using TaskTypes = typename svtkm::cont::DeviceTaskTypes<DeviceAdapter>;
  auto task2 = TaskTypes::MakeTask(worklet, invocation2, svtkm::Id());

  ScheduleTaskStrided<decltype(task2)><<<32, 256>>>(task2, 2, 3);
  hipDeviceSynchronize();
  input.SyncControlArray();
  output.SyncControlArray();

  SVTKM_TEST_ASSERT(inputTestValues[2] == 6, "Input value changed.");
  SVTKM_TEST_ASSERT(output.GetPortalConstControl().Get(2) == inputTestValues[2] + 200 + 30 * 2,
                   "Output value not set right.");
}

template <typename DeviceAdapter>
void TestErrorFunctorInvoke()
{
  std::cout << "Testing invoke with an error raised in the worklet." << std::endl;

  svtkm::Id inputTestValue = 5;
  svtkm::Id outputTestValue = static_cast<svtkm::Id>(0xDEADDEAD);

  svtkm::cont::ArrayHandle<svtkm::Id> input = svtkm::cont::make_ArrayHandle(&inputTestValue, 1);
  svtkm::cont::ArrayHandle<svtkm::Id> output = svtkm::cont::make_ArrayHandle(&outputTestValue, 1);

  svtkm::internal::FunctionInterface<void(TestExecObject, TestExecObject)> execObjects =
    svtkm::internal::make_FunctionInterface<void>(
      TestExecObject(input.PrepareForInPlace(DeviceAdapter())),
      TestExecObject(output.PrepareForInPlace(DeviceAdapter())));

  using TaskStrided1 =
    svtkm::exec::cuda::internal::TaskStrided1D<TestWorkletErrorProxy, InvocationType1>;
  TestWorkletErrorProxy worklet;
  InvocationType1 invocation(execObjects);

  using TaskTypes = typename svtkm::cont::DeviceTaskTypes<DeviceAdapter>;
  using Algorithm = svtkm::cont::DeviceAdapterAlgorithm<DeviceAdapter>;

  auto task = TaskTypes::MakeTask(worklet, invocation, svtkm::Id());

  auto errorArray = Algorithm::GetPinnedErrorArray();
  svtkm::exec::internal::ErrorMessageBuffer errorMessage(errorArray.DevicePtr, errorArray.Size);
  task.SetErrorMessageBuffer(errorMessage);

  ScheduleTaskStrided<decltype(task)><<<32, 256>>>(task, 1, 2);
  hipDeviceSynchronize();

  SVTKM_TEST_ASSERT(errorMessage.IsErrorRaised(), "Error not raised correctly.");
  SVTKM_TEST_ASSERT(errorArray.HostPtr == std::string(ERROR_MESSAGE), "Got wrong error message.");
}

template <typename DeviceAdapter>
void TestTaskStrided()
{
  TestNormalFunctorInvoke<DeviceAdapter>();
  TestErrorFunctorInvoke<DeviceAdapter>();
}

} // anonymous namespace

int UnitTestTaskStrided(int argc, char* argv[])
{
  return svtkm::testing::Testing::Run(TestTaskStrided<svtkm::cont::DeviceAdapterTagCuda>, argc, argv);
}

#if defined(__NVCC__) && defined(__CUDACC_VER_MAJOR__)
#pragma pop
#endif

#if defined(SVTKM_MSVC)
#pragma warning(pop)
#endif
