//============================================================================
//  Copyright (c) Kitware, Inc.
//  All rights reserved.
//  See LICENSE.txt for details.
//
//  This software is distributed WITHOUT ANY WARRANTY; without even
//  the implied warranty of MERCHANTABILITY or FITNESS FOR A PARTICULAR
//  PURPOSE.  See the above copyright notice for more information.
//============================================================================
#include <svtkm/cont/cuda/internal/DeviceAdapterTimerImplementationCuda.h>

#include <svtkm/Types.h>
#include <svtkm/cont/cuda/ErrorCuda.h>

#include <hip/hip_runtime.h>

namespace svtkm
{
namespace cont
{

DeviceAdapterTimerImplementation<
  svtkm::cont::DeviceAdapterTagCuda>::DeviceAdapterTimerImplementation()
{
  SVTKM_CUDA_CALL(hipEventCreate(&this->StartEvent));
  SVTKM_CUDA_CALL(hipEventCreate(&this->StopEvent));
  this->Reset();
}

DeviceAdapterTimerImplementation<
  svtkm::cont::DeviceAdapterTagCuda>::~DeviceAdapterTimerImplementation()
{
  // These aren't wrapped in SVTKM_CUDA_CALL because we can't throw errors
  // from destructors. We're relying on hipGetLastError in the
  // SVTKM_CUDA_CHECK_ASYNCHRONOUS_ERROR catching any issues from these calls
  // later.
  hipEventDestroy(this->StartEvent);
  hipEventDestroy(this->StopEvent);
}

void DeviceAdapterTimerImplementation<svtkm::cont::DeviceAdapterTagCuda>::Reset()
{
  this->StartReady = false;
  this->StopReady = false;
}

void DeviceAdapterTimerImplementation<svtkm::cont::DeviceAdapterTagCuda>::Start()
{
  SVTKM_CUDA_CALL(hipEventRecord(this->StartEvent, hipStreamPerThread));
  this->StartReady = true;
}

void DeviceAdapterTimerImplementation<svtkm::cont::DeviceAdapterTagCuda>::Stop()
{
  SVTKM_CUDA_CALL(hipEventRecord(this->StopEvent, hipStreamPerThread));
  SVTKM_CUDA_CALL(hipEventSynchronize(this->StopEvent));
  this->StopReady = true;
}

bool DeviceAdapterTimerImplementation<svtkm::cont::DeviceAdapterTagCuda>::Started() const
{
  return this->StartReady;
}

bool DeviceAdapterTimerImplementation<svtkm::cont::DeviceAdapterTagCuda>::Stopped() const
{
  return this->StopReady;
}

// Callbacks without a mandated order(in independent streams) execute in undefined
// order and maybe serialized. So Instead CudaEventQuery is used here.
// Ref link: https://docs.nvidia.com/cuda/cuda-driver-api/group__CUDA__STREAM.html
bool DeviceAdapterTimerImplementation<svtkm::cont::DeviceAdapterTagCuda>::Ready() const
{
  if (hipEventQuery(this->StopEvent) == hipSuccess)
  {
    return true;
  }
  return false;
}


svtkm::Float64 DeviceAdapterTimerImplementation<svtkm::cont::DeviceAdapterTagCuda>::GetElapsedTime()
  const
{
  assert(this->StartReady);
  if (!this->StartReady)
  {
    SVTKM_LOG_F(svtkm::cont::LogLevel::Error,
               "Start() function should be called first then trying to call GetElapsedTime().");
    return 0;
  }
  if (!this->StopReady)
  {
    // Stop was not called, so we have to insert a new event into the stream
    SVTKM_CUDA_CALL(hipEventRecord(this->StopEvent, hipStreamPerThread));
    SVTKM_CUDA_CALL(hipEventSynchronize(this->StopEvent));
  }

  float elapsedTimeMilliseconds;
  SVTKM_CUDA_CALL(hipEventElapsedTime(&elapsedTimeMilliseconds, this->StartEvent, this->StopEvent));
  // Reset Stop flag to its original state
  return static_cast<svtkm::Float64>(0.001f * elapsedTimeMilliseconds);
}
}
} // namespace svtkm::cont
