//============================================================================
//  Copyright (c) Kitware, Inc.
//  All rights reserved.
//  See LICENSE.txt for details.
//
//  This software is distributed WITHOUT ANY WARRANTY; without even
//  the implied warranty of MERCHANTABILITY or FITNESS FOR A PARTICULAR
//  PURPOSE.  See the above copyright notice for more information.
//============================================================================

#include <svtkm/cont/cuda/internal/testing/Testing.h>

#include <svtkm/cont/ArrayHandle.h>
#include <svtkm/cont/RuntimeDeviceTracker.h>

#include <svtkm/cont/cuda/DeviceAdapterCuda.h>
#include <svtkm/cont/cuda/ErrorCuda.h>

#include <svtkm/cont/cuda/internal/CudaAllocator.h>
#include <svtkm/cont/cuda/internal/testing/Testing.h>

#include <hip/hip_runtime.h>

using svtkm::cont::cuda::internal::CudaAllocator;

namespace
{

template <typename ValueType>
ValueType* AllocateManagedPointer(svtkm::Id numValues)
{
  void* result;
  SVTKM_CUDA_CALL(hipMallocManaged(&result, static_cast<size_t>(numValues) * sizeof(ValueType)));
  // Some sanity checks:
  SVTKM_TEST_ASSERT(CudaAllocator::IsDevicePointer(result),
                   "Allocated pointer is not a device pointer.");
  SVTKM_TEST_ASSERT(CudaAllocator::IsManagedPointer(result), "Allocated pointer is not managed.");
  return static_cast<ValueType*>(result);
}

template <typename ValueType>
ValueType* AllocateDevicePointer(svtkm::Id numValues)
{
  void* result;
  SVTKM_CUDA_CALL(hipMalloc(&result, static_cast<size_t>(numValues) * sizeof(ValueType)));
  // Some sanity checks:
  SVTKM_TEST_ASSERT(CudaAllocator::IsDevicePointer(result),
                   "Allocated pointer is not a device pointer.");
  SVTKM_TEST_ASSERT(!CudaAllocator::IsManagedPointer(result), "Allocated pointer is managed.");
  return static_cast<ValueType*>(result);
}

template <typename ValueType>
svtkm::cont::ArrayHandle<ValueType> CreateArrayHandle(svtkm::Id numValues, bool managed)
{
  ValueType* ptr = managed ? AllocateManagedPointer<ValueType>(numValues)
                           : AllocateDevicePointer<ValueType>(numValues);
  return svtkm::cont::make_ArrayHandle(ptr, numValues);
}

template <typename ValueType>
void TestPrepareForInput(bool managed)
{
  svtkm::cont::ArrayHandle<ValueType> handle = CreateArrayHandle<ValueType>(32, managed);
  handle.PrepareForInput(svtkm::cont::DeviceAdapterTagCuda());

  auto lock = handle.Internals->GetLock();
  void* contArray = handle.Internals->Internals->GetControlArray(lock)->GetBasePointer();
  void* execArray = handle.Internals->Internals->GetExecutionArray(lock);
  SVTKM_TEST_ASSERT(contArray != nullptr, "No control array after PrepareForInput.");
  SVTKM_TEST_ASSERT(execArray != nullptr, "No execution array after PrepareForInput.");
  SVTKM_TEST_ASSERT(CudaAllocator::IsDevicePointer(execArray),
                   "PrepareForInput execution array not device pointer.");
  SVTKM_TEST_ASSERT(CudaAllocator::IsDevicePointer(contArray),
                   "PrepareForInput control array not device pointer.");
  if (managed)
  {
    SVTKM_TEST_ASSERT(CudaAllocator::IsManagedPointer(execArray),
                     "PrepareForInput execution array unmanaged.");
    SVTKM_TEST_ASSERT(CudaAllocator::IsManagedPointer(contArray),
                     "PrepareForInput control array unmanaged.");
  }
  SVTKM_TEST_ASSERT(execArray == contArray, "PrepareForInput managed arrays not shared.");
}

template <typename ValueType>
void TestPrepareForInPlace(bool managed)
{
  svtkm::cont::ArrayHandle<ValueType> handle = CreateArrayHandle<ValueType>(32, managed);
  handle.PrepareForInPlace(svtkm::cont::DeviceAdapterTagCuda());

  auto lock = handle.Internals->GetLock();
  void* contArray = handle.Internals->Internals->GetControlArray(lock)->GetBasePointer();
  void* execArray = handle.Internals->Internals->GetExecutionArray(lock);
  SVTKM_TEST_ASSERT(contArray != nullptr, "No control array after PrepareForInPlace.");
  SVTKM_TEST_ASSERT(execArray != nullptr, "No execution array after PrepareForInPlace.");
  SVTKM_TEST_ASSERT(CudaAllocator::IsDevicePointer(execArray),
                   "PrepareForInPlace execution array not device pointer.");
  SVTKM_TEST_ASSERT(CudaAllocator::IsDevicePointer(contArray),
                   "PrepareForInPlace control array not device pointer.");
  if (managed)
  {
    SVTKM_TEST_ASSERT(CudaAllocator::IsManagedPointer(execArray),
                     "PrepareForInPlace execution array unmanaged.");
    SVTKM_TEST_ASSERT(CudaAllocator::IsManagedPointer(contArray),
                     "PrepareForInPlace control array unmanaged.");
  }
  SVTKM_TEST_ASSERT(execArray == contArray, "PrepareForInPlace managed arrays not shared.");
}

template <typename ValueType>
void TestPrepareForOutput(bool managed)
{
  // Should reuse a managed control pointer if buffer is large enough.
  svtkm::cont::ArrayHandle<ValueType> handle = CreateArrayHandle<ValueType>(32, managed);
  handle.PrepareForOutput(32, svtkm::cont::DeviceAdapterTagCuda());

  auto lock = handle.Internals->GetLock();
  void* contArray = handle.Internals->Internals->GetControlArray(lock)->GetBasePointer();
  void* execArray = handle.Internals->Internals->GetExecutionArray(lock);
  SVTKM_TEST_ASSERT(contArray != nullptr, "No control array after PrepareForOutput.");
  SVTKM_TEST_ASSERT(execArray != nullptr, "No execution array after PrepareForOutput.");
  SVTKM_TEST_ASSERT(CudaAllocator::IsDevicePointer(execArray),
                   "PrepareForOutput execution array not device pointer.");
  SVTKM_TEST_ASSERT(CudaAllocator::IsDevicePointer(contArray),
                   "PrepareForOutput control array not device pointer.");
  if (managed)
  {
    SVTKM_TEST_ASSERT(CudaAllocator::IsManagedPointer(execArray),
                     "PrepareForOutput execution array unmanaged.");
    SVTKM_TEST_ASSERT(CudaAllocator::IsManagedPointer(contArray),
                     "PrepareForOutput control array unmanaged.");
  }
  SVTKM_TEST_ASSERT(execArray == contArray, "PrepareForOutput managed arrays not shared.");
}

template <typename ValueType>
void TestReleaseResourcesExecution(bool managed)
{
  svtkm::cont::ArrayHandle<ValueType> handle = CreateArrayHandle<ValueType>(32, managed);
  handle.PrepareForInput(svtkm::cont::DeviceAdapterTagCuda());

  void* origArray;
  {
    auto lock = handle.Internals->GetLock();
    origArray = handle.Internals->Internals->GetExecutionArray(lock);
  }

  handle.ReleaseResourcesExecution();

  auto lock = handle.Internals->GetLock();
  void* contArray = handle.Internals->Internals->GetControlArray(lock)->GetBasePointer();
  void* execArray = handle.Internals->Internals->GetExecutionArray(lock);

  SVTKM_TEST_ASSERT(contArray != nullptr, "No control array after ReleaseResourcesExecution.");
  SVTKM_TEST_ASSERT(execArray == nullptr,
                   "Execution array not cleared after ReleaseResourcesExecution.");
  SVTKM_TEST_ASSERT(CudaAllocator::IsDevicePointer(contArray),
                   "ReleaseResourcesExecution control array not device pointer.");
  if (managed)
  {
    SVTKM_TEST_ASSERT(CudaAllocator::IsManagedPointer(contArray),
                     "ReleaseResourcesExecution control array unmanaged.");
  }
  SVTKM_TEST_ASSERT(origArray == contArray,
                   "Control array changed after ReleaseResourcesExecution.");
}

template <typename ValueType>
void TestRoundTrip(bool managed)
{
  svtkm::cont::ArrayHandle<ValueType> handle = CreateArrayHandle<ValueType>(32, managed);
  handle.PrepareForOutput(32, svtkm::cont::DeviceAdapterTagCuda());

  void* origContArray;
  {
    auto lock = handle.Internals->GetLock();
    origContArray = handle.Internals->Internals->GetControlArray(lock)->GetBasePointer();
  }
  {
    auto lock = handle.Internals->GetLock();
    void* contArray = handle.Internals->Internals->GetControlArray(lock)->GetBasePointer();
    void* execArray = handle.Internals->Internals->GetExecutionArray(lock);
    SVTKM_TEST_ASSERT(contArray != nullptr, "No control array after PrepareForOutput.");
    SVTKM_TEST_ASSERT(execArray != nullptr, "No execution array after PrepareForOutput.");
    SVTKM_TEST_ASSERT(CudaAllocator::IsDevicePointer(execArray),
                     "PrepareForOutput execution array not device pointer.");
    SVTKM_TEST_ASSERT(CudaAllocator::IsDevicePointer(contArray),
                     "PrepareForOutput control array not device pointer.");
    if (managed)
    {
      SVTKM_TEST_ASSERT(CudaAllocator::IsManagedPointer(execArray),
                       "PrepareForOutput execution array unmanaged.");
      SVTKM_TEST_ASSERT(CudaAllocator::IsManagedPointer(contArray),
                       "PrepareForOutput control array unmanaged.");
    }
    SVTKM_TEST_ASSERT(execArray == contArray, "PrepareForOutput managed arrays not shared.");
  }

  try
  {
    handle.GetPortalControl();
  }
  catch (svtkm::cont::ErrorBadValue&)
  {
    if (managed)
    {
      throw; // Exception is unexpected
    }

    // If !managed, this exception is intentional to indicate that the control
    // array is a non-managed device pointer, and thus unaccessible from the
    // control environment. Return because we've already validated correct
    // behavior by catching this exception.
    return;
  }

  if (!managed)
  {
    SVTKM_TEST_FAIL("Expected exception not thrown.");
  }

  {
    auto lock = handle.Internals->GetLock();
    void* contArray = handle.Internals->Internals->GetControlArray(lock)->GetBasePointer();
    void* execArray = handle.Internals->Internals->GetExecutionArray(lock);
    SVTKM_TEST_ASSERT(contArray != nullptr, "No control array after GetPortalConst.");
    SVTKM_TEST_ASSERT(execArray == nullptr, "Execution array not cleared after GetPortalConst.");
    SVTKM_TEST_ASSERT(CudaAllocator::IsDevicePointer(contArray),
                     "GetPortalConst control array not device pointer.");
    SVTKM_TEST_ASSERT(origContArray == contArray, "GetPortalConst changed control array.");
  }
}

template <typename ValueType>
void DoTests()
{
  TestPrepareForInput<ValueType>(false);
  TestPrepareForInPlace<ValueType>(false);
  TestPrepareForOutput<ValueType>(false);
  TestReleaseResourcesExecution<ValueType>(false);
  TestRoundTrip<ValueType>(false);


  // If this device does not support managed memory, skip the managed tests.
  if (!CudaAllocator::UsingManagedMemory())
  {
    std::cerr << "Skipping some tests -- device does not support managed memory.\n";
  }
  else
  {
    TestPrepareForInput<ValueType>(true);
    TestPrepareForInPlace<ValueType>(true);
    TestPrepareForOutput<ValueType>(true);
    TestReleaseResourcesExecution<ValueType>(true);
    TestRoundTrip<ValueType>(true);
  }
}

struct ArgToTemplateType
{
  template <typename ValueType>
  void operator()(ValueType) const
  {
    DoTests<ValueType>();
  }
};

void Launch()
{
  using Types = svtkm::List<svtkm::UInt8,
                           svtkm::Vec<svtkm::UInt8, 3>,
                           svtkm::Float32,
                           svtkm::Vec<svtkm::Float32, 4>,
                           svtkm::Float64,
                           svtkm::Vec<svtkm::Float64, 4>>;
  svtkm::testing::Testing::TryTypes(ArgToTemplateType(), Types());
}

} // end anon namespace

int UnitTestCudaShareUserProvidedManagedMemory(int argc, char* argv[])
{
  auto& tracker = svtkm::cont::GetRuntimeDeviceTracker();
  tracker.ForceDevice(svtkm::cont::DeviceAdapterTagCuda{});
  int ret = svtkm::cont::testing::Testing::Run(Launch, argc, argv);
  return svtkm::cont::cuda::internal::Testing::CheckCudaBeforeExit(ret);
}
